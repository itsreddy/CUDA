#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <stdio.h>
//#include "UnifiedMathCUDA.cuh"
#include "helper_functions.h"
#include <Windows.h>

hipError_t addWithCuda(int *c, unsigned int size);

__global__ void addKernel(int *c)
{
	int i = threadIdx.x;
	double phi = 1.618033988749895;  // Golden Ratio
		c[i] = round((pow(phi, i) - pow(-phi, -i)) / sqrt((double)5));
}

int main()
{
	__int64 ctr1 = 0, ctr2 = 0, freq = 0;
	const int arraySize = 1000;
	int NoThreads = 8;
	//const int a[arraySize] = { 1, 2, 3, 4, 5 };
	//const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	QueryPerformanceCounter((LARGE_INTEGER *)&ctr1);    //profiling

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	QueryPerformanceCounter((LARGE_INTEGER *)&ctr2);
	QueryPerformanceFrequency((LARGE_INTEGER *)&freq);
	printf("Time taken (In seconds) : %f\n", ((ctr2 - ctr1) * 1.0 / freq));


	printf("Sequence size: %d\nFibonacci sequence: ", arraySize);
	/*	for (int k = 0; k < arraySize; k++)
		printf("%d ", c[k]);
	
	printf("\n");
	*/
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);

	return cudaStatus;
}
